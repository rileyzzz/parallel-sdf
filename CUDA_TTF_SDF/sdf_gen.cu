#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sdf_gen.cuh"

#include <memory.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

CUDA_SDF::SDFGenerationContext::~SDFGenerationContext()
{
	hipFree(data);
	hipFree(sdf_data);
	//hipFree(out_data);
	delete[] out_data;
}

void CUDA_SDF::SDFGenerationContext::CopyImage(unsigned char* img_data)
{
	hipFree(data);
	hipFree(sdf_data);
	//hipFree(out_data);
	delete[] out_data;

	size_t img_size = (size_t)width * (size_t)height * (size_t)numComponents;

	hipMallocManaged(&data, img_size);
	hipMallocManaged(&sdf_data, img_size * sizeof(float));
	//hipMallocManaged(&out_data, img_size);
	out_data = new unsigned char[img_size];
	memcpy(data, img_data, img_size);
}

template <typename type>
inline __host__ __device__ type clamp(type d, type min, type max)
{
	const type t = d < min ? min : d;
	return t > max ? max : t;
}

inline __host__ __device__ float distance(int x1, int y1, int x2, int y2)
{
	float x = (float)x2 - (float)x1;
	float y = (float)y2 - (float)y1;

	return sqrt(x * x + y * y);
}

#define max(a,b) (((a) > (b)) ? (a) : (b))
#define min(a,b) (((a) < (b)) ? (a) : (b))

__global__ void sdf_calc(int width, int height, unsigned char* data, float* out)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int stride = blockDim.x;

#define SAMPLE(_x, _y) data[_y * stride + _x]
#define WRITE(_x, _y, val) out[_y * stride + _x] = val

	bool isBaseSample = SAMPLE(x, y) > 0x00;

	const int maxRadius = 16;

	float dist = distance(0, 0, width, height);
	bool foundSample = false;

	for (int ringSize = 1; ringSize <= maxRadius; ringSize++)
	{
		int fromX	= clamp(x - ringSize, 0, width - 1);
		int toX		= clamp(x + ringSize, 0, width - 1);
		int fromY	= clamp(y - ringSize, 0, height - 1);
		int toY		= clamp(y + ringSize, 0, height - 1);

		// ***
		// ---
		// ---
		for (int i = fromX; i <= toX; i++)
		{
			int uv_x = i;
			int uv_y = fromY;

			unsigned char sample = SAMPLE(uv_x, uv_y);
			if (sample > 0 && !isBaseSample || sample == 0 && isBaseSample)
			{
				dist = min(dist, distance(x, y, uv_x, uv_y));
				foundSample = true;
			}
		}

		// ---
		// ---
		// ***
		for (int i = fromX; i <= toX; i++)
		{
			int uv_x = i;
			int uv_y = toY;

			unsigned char sample = SAMPLE(uv_x, uv_y);
			if (sample > 0 && !isBaseSample || sample == 0 && isBaseSample)
			{
				dist = min(dist, distance(x, y, uv_x, uv_y));
				foundSample = true;
			}
		}

		// ---
		// *--
		// ---
		for (int i = fromY + 1; i < toY; i++)
		{
			int uv_x = fromX;
			int uv_y = i;

			unsigned char sample = SAMPLE(uv_x, uv_y);
			if (sample > 0 && !isBaseSample || sample == 0 && isBaseSample)
			{
				dist = min(dist, distance(x, y, uv_x, uv_y));
				foundSample = true;
			}
		}

		// ---
		// --*
		// ---
		for (int i = fromY + 1; i < toY; i++)
		{
			int uv_x = toX;
			int uv_y = i;

			unsigned char sample = SAMPLE(uv_x, uv_y);
			if (sample > 0 && !isBaseSample || sample == 0 && isBaseSample)
			{
				dist = min(dist, distance(x, y, uv_x, uv_y));
				foundSample = true;
			}
		}

		//if (foundSample)
		//{
		//	break;
		//}
	}

	//dist = 1.0f - (dist / (float)maxRadius);
	//dist = clamp(dist, 0.0f, 1.0f);
	if (!isBaseSample)
		dist = -dist;

	WRITE(x, y, dist);

#undef SAMPLE
#undef WRITE
}

void CUDA_SDF::GenerateSDF(SDFGenerationContext& ctx)
{
	printf("Generating signed distance field.\n");

	int numThreads = ctx.width * ctx.height;
	int blockSize = ctx.width;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;
	//add << <numBlocks, blockSize >> > (N, x, y);

	sdf_calc << < numBlocks, blockSize >> > (ctx.width, ctx.height, ctx.data, ctx.sdf_data);

	hipDeviceSynchronize();

	float maxDist = distance(0, 0, ctx.width, ctx.height);
	float sdf_min = maxDist;
	float sdf_max = -sdf_min;

	for (int i = 0; i < ctx.width * ctx.height; i++)
	{
		const float& val = ctx.sdf_data[i];

		//if the value is not a max distance sample
		if (val > -maxDist && val < maxDist)
		{
			sdf_min = min(sdf_min, val);
			sdf_max = max(sdf_max, val);
		}
	}

	printf("SDF min %f max %f\n", sdf_min, sdf_max);

	float sdf_range = sdf_max - sdf_min;
	for (int i = 0; i < ctx.width * ctx.height; i++)
	{
		const float& val = ctx.sdf_data[i];
		float normalized_val = (val - sdf_min) / sdf_range;
		//float normalized_val = val / sdf_range * 0.5f + 0.5f;
		normalized_val = clamp(normalized_val, 0.0f, 1.0f);
		ctx.out_data[i] = (unsigned char)(normalized_val * 255.0f);
	}
}

